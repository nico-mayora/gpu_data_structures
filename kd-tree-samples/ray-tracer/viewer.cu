#include "viewer.cuh"
#include "../common/data/world.cuh"
#include "cuda/pathTracer.cuh"

extern "C" char pathTracer_ptx[];

Viewer::Viewer(const World *world) {
    context = owlContextCreate(nullptr, 1);
    owlContextSetRayTypeCount(context, RAY_TYPES_COUNT);
    OWLModule module = owlModuleCreate(context, pathTracer_ptx);

    OWLVarDecl triangles_geom_vars[] = {
        { "material", OWL_RAW_POINTER, OWL_OFFSETOF(TrianglesGeomData,material)},
        { "vertex", OWL_BUFPTR, OWL_OFFSETOF(TrianglesGeomData,vertex)},
        { "index",  OWL_BUFPTR, OWL_OFFSETOF(TrianglesGeomData,index)},
        { "normal",  OWL_BUFPTR, OWL_OFFSETOF(TrianglesGeomData,normal)},
        { "faceted", OWL_BOOL, OWL_OFFSETOF(TrianglesGeomData, faceted)},
        { nullptr /* Sentinel to mark end-of-list */}
    };

    OWLGeomType triangles_geom_type
        = owlGeomTypeCreate(context,
                          OWL_TRIANGLES,
                          sizeof(TrianglesGeomData),
                          triangles_geom_vars,-1);
    owlGeomTypeSetClosestHit(triangles_geom_type,PRIMARY,
                             module,"TriangleMesh");
    owlGeomTypeSetClosestHit(triangles_geom_type, SHADOW, module,"shadow");

    std::cout << "Building geometries...\n";

    // Upload meshes to GPU
    std::vector<OWLGeom> geometries;
    for (const auto model : world->models) {
        const auto mesh = model->mesh;
        OWLBuffer vertex_buffer
            = owlDeviceBufferCreate(context, OWL_FLOAT3, mesh->vertices.size(), mesh->vertices.data());
        OWLBuffer normal_buffer
            = owlDeviceBufferCreate(context, OWL_FLOAT3, mesh->normals.size(), mesh->normals.data());
        OWLBuffer index_buffer
            = owlDeviceBufferCreate(context,OWL_INT3,mesh->indices.size(), mesh->indices.data());

        OWLGeom triangles_geom
            = owlGeomCreate(context, triangles_geom_type);

        owlTrianglesSetVertices(triangles_geom,vertex_buffer,
                                mesh->vertices.size(),sizeof(owl::vec3f),0);
        owlTrianglesSetIndices(triangles_geom,index_buffer,
                               mesh->indices.size(),sizeof(owl::vec3i),0);

        owlGeomSetBuffer(triangles_geom,"vertex", vertex_buffer);
        owlGeomSetBuffer(triangles_geom,"index", index_buffer);
        owlGeomSetBuffer(triangles_geom,"normal", normal_buffer);
        owlGeomSet1b(triangles_geom, "faceted", mesh->faceted);

        // Copy material to device memory.
        Material *mat_ptr;
        hipMalloc(reinterpret_cast<void**>(&mat_ptr),sizeof(Material));
        hipMemcpy(mat_ptr, model->material, sizeof(Material), hipMemcpyHostToDevice);
        owlGeomSetPointer(triangles_geom, "material", mat_ptr);

        geometries.emplace_back(triangles_geom);
    }

    OWLGroup triangles_group
        = owlTrianglesGeomGroupCreate(context,geometries.size(),geometries.data());
    owlGroupBuildAccel(triangles_group);
    OWLGroup owl_world
        = owlInstanceGroupCreate(context,1);
    owlInstanceGroupSetChild(owl_world,0,triangles_group);
    owlGroupBuildAccel(owl_world);

    // Miss program
    OWLVarDecl missProgVars[] =
    {
        { "sky_colour", OWL_FLOAT3, OWL_OFFSETOF(MissProgData, sky_colour)},
        { /* sentinel to mark end of list */ }
    };
    OWLMissProg missProg
      = owlMissProgCreate(context,module,"miss",sizeof(MissProgData),
                          missProgVars,-1);
    owlMissProgSet3f(missProg,"sky_colour",owl3f{.1f,.01f,.2f});

    owlMissProgCreate(context, module,"shadow",0,nullptr,-1);

    OWLVarDecl rayGenVars[] = {
        { "fbPtr",         OWL_RAW_POINTER, OWL_OFFSETOF(RayGenData,fbPtr)},
        { "depth", OWL_INT, OWL_OFFSETOF(RayGenData,depth)},
        { "pixel_samples", OWL_INT, OWL_OFFSETOF(RayGenData,pixel_samples)},
        { "num_diffuse_scattered", OWL_INT, OWL_OFFSETOF(RayGenData,num_diffuse_scattered)},
        { "photon_map", OWL_RAW_POINTER, OWL_OFFSETOF(RayGenData,photon_map)},
        { "num_photons", OWL_INT, OWL_OFFSETOF(RayGenData,num_photons)},
        { "resolution", OWL_INT2, OWL_OFFSETOF(RayGenData,resolution)},
        { "world",         OWL_GROUP,  OWL_OFFSETOF(RayGenData,world)},
        { "camera.pos",    OWL_FLOAT3, OWL_OFFSETOF(RayGenData,camera.pos)},
        { "camera.dir_00", OWL_FLOAT3, OWL_OFFSETOF(RayGenData,camera.dir_00)},
        { "camera.dir_dv", OWL_FLOAT3, OWL_OFFSETOF(RayGenData,camera.dir_dv)},
        { "camera.dir_du", OWL_FLOAT3, OWL_OFFSETOF(RayGenData,camera.dir_du)},
        { "scene_light", OWL_BUFPTR, OWL_OFFSETOF(RayGenData,scene_light)},
        { /* sentinel to mark end of list */ },
    };

    rayGen
        = owlRayGenCreate(context,module,"ptRayGen", sizeof(RayGenData), rayGenVars,-1);
    owlRayGenSetGroup(rayGen,"world", owl_world);

    auto scene_light_buf = owlDeviceBufferCreate(context, OWL_USER_TYPE(PointLight), 1, world->scene_light);
    owlRayGenSetBuffer(rayGen,"scene_light", scene_light_buf);

    // Initialise Viewer camera with params from scene description.
    camera.setOrientation(world->cam->lookFrom,
                          world->cam->lookAt,
                          world->cam->up,
                          world->cam->image.fov);

    // Set RayGen constant attributes
    owlRayGenSet1i(rayGen, "pixel_samples", world->cam->image.pixel_samples);
    owlRayGenSet1i(rayGen, "num_diffuse_scattered", world->cam->image.num_diffuse_scattered);
    owlRayGenSetPointer(rayGen, "photon_map", world->photon_map);
    owlRayGenSet1i(rayGen, "num_photons", world->num_photons);
    owlRayGenSet1i(rayGen, "depth", world->cam->image.depth);
    owlRayGenSet2i(rayGen, "resolution", reinterpret_cast<const owl2i&>(world->cam->image.resolution));
    setWindowSize(world->cam->image.resolution);

    owlBuildPrograms(context);
    owlBuildPipeline(context);
    owlBuildSBT(context);
}

void Viewer::render()
{
    if (sbtDirty) {
        owlBuildSBT(context);
        sbtDirty = false;
    }
    std::cout << "Launching...\n";
    owlRayGenLaunch2D(rayGen,fbSize.x,fbSize.y);
}

void Viewer::resize(const owl::vec2i &newSize)
{
    OWLViewer::resize(newSize);
    cameraChanged();
}

void Viewer::cameraChanged()
{
    const owl::vec3f lookFrom = camera.getFrom();
    const owl::vec3f lookAt = camera.getAt();
    const owl::vec3f lookUp = camera.getUp();

    const float cosFovy = camera.getCosFovy();
    // ----------- compute variable values  ------------------
    owl::vec3f camera_pos = lookFrom;
    owl::vec3f camera_d00
      = normalize(lookAt-lookFrom);
    float aspect = fbSize.x / float(fbSize.y);
    owl::vec3f camera_ddu
      = cosFovy * aspect * normalize(cross(camera_d00,lookUp));
    owl::vec3f camera_ddv
      = cosFovy * normalize(cross(camera_ddu,camera_d00));
    camera_d00 -= 0.5f * camera_ddu;
    camera_d00 -= 0.5f * camera_ddv;

    // ----------- set variables  ----------------------------
    owlRayGenSet1ul   (rayGen,"fbPtr",        reinterpret_cast<uint64_t>(fbPointer));
    owlRayGenSet2i    (rayGen,"resolution",   reinterpret_cast<const owl2i&>(fbSize));
    owlRayGenSet3f    (rayGen,"camera.pos",   reinterpret_cast<const owl3f&>(camera_pos));
    owlRayGenSet3f    (rayGen,"camera.dir_00",reinterpret_cast<const owl3f&>(camera_d00));
    owlRayGenSet3f    (rayGen,"camera.dir_du",reinterpret_cast<const owl3f&>(camera_ddu));
    owlRayGenSet3f    (rayGen,"camera.dir_dv",reinterpret_cast<const owl3f&>(camera_ddv));
    sbtDirty = true;
}