#include "builder.cuh"
#include "queries.cuh"
#include <iostream>

static std::string print_coords(const Point p) {
    return "("
        + std::to_string(p.coords[0]) + ", "
        + std::to_string(p.coords[1]) + /* ", "
        + std::to_string(p.coords[2]) + */")";
}

static void print_point_buffer(const Point *points, const int N) {
    for (int i = 0; i < N; ++i) {
        std::cout << "Coords: " << print_coords(points[i]) << " Payload: '" << points[i].payload << "'\n";
    }
}

static int nodes_in_level(const int l) {
    return 1 << l;
}
static int num_levels(const int N) {
    return 32 - std::countl_zero(static_cast<uint32_t>(N));
}

static void print_kd_tree(const Point *points, const int N) {
    const int maxLvl = num_levels(N);
    int offset = 0;
    for (int l = 0; l < maxLvl; ++l) {
        const int inCurrLevel = nodes_in_level(l);
        std::cout << "Level " << l << ": ";
        for (int k = 0; k < inCurrLevel; ++k) {
            const int idx =  offset + k;
            if (idx >= N) break;
            std::cout << "[" << points[idx].payload << ": " << print_coords(points[idx]) << "]";
        }
        std::cout << "\n";
        offset += inCurrLevel;
    }
}


int main() {
    std::vector points_vec = {
        Point { 'a', {10., 15.} },
        Point { 'b', {46., 63.} },
        Point { 'c', {68., 21.} },
        Point { 'd', {40., 33.} },
        Point { 'e', {25., 54.} },
        Point { 'f', {15., 43.} },
        Point { 'g', {44., 58.} },
        Point { 'h', {45., 40.} },
        Point { 'i', {62., 69.} },
        Point { 'j', {53., 67.} }
    };

    const auto points = &points_vec[0];
    const int N = points_vec.size();

    std::cout << "Points is: \n";
    print_point_buffer(points, N);

    // Create device buffer for points
    Point *d_points;
    const size_t device_buffer_size = N * sizeof(points[0]);
    hipMalloc(&d_points, device_buffer_size);
    hipMemcpy(d_points, points, device_buffer_size, hipMemcpyHostToDevice);

    // Build the Kd-tree
    std::cout << "Building KDTREE... \n";
    build_kd_tree(d_points, N);

    // Copy it back to the host
    const auto host_points = new Point[N];
    hipMemcpy(host_points, d_points, device_buffer_size, hipMemcpyDeviceToHost);

    // Print it in level-order
    std::cout << "KDTREE is: \n";
    print_kd_tree(host_points, N);

    // Clean up
    hipFree(points);
    delete[] host_points;
}
